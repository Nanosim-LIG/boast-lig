
#include <hip/hip_runtime.h>
#ifndef INDEX2
#define INDEX2(xsize,x,y) x + (y)*xsize
#endif
#ifndef INDEX3
#define INDEX3(xsize,ysize,x,y,z) x + xsize*(y + ysize*z)
#endif
#ifndef INDEX4
#define INDEX4(xsize,ysize,zsize,x,y,z,i) x + xsize*(y + ysize*(z + zsize*i))
#endif
#ifndef INDEX5
#define INDEX5(xsize,ysize,zsize,isize,x,y,z,i,j) x + xsize*(y + ysize*(z + zsize*(i + isize*(j))))
#endif
#ifndef NDIM
#define NDIM 3
#endif
#ifndef NGLLX
#define NGLLX 5
#endif
#ifndef NGLL2
#define NGLL2 25
#endif
#ifndef NGLL3
#define NGLL3 125
#endif
#ifndef NGLL3_PADDED
#define NGLL3_PADDED 128
#endif
#ifndef N_SLS
#define N_SLS 3
#endif
#ifndef IREGION_CRUST_MANTLE
#define IREGION_CRUST_MANTLE 1
#endif
#ifndef IREGION_INNER_CORE
#define IREGION_INNER_CORE 3
#endif
#ifndef IFLAG_IN_FICTITIOUS_CUBE
#define IFLAG_IN_FICTITIOUS_CUBE 11
#endif
#ifndef R_EARTH_KM
#define R_EARTH_KM 6371.0f
#endif
#ifndef COLORING_MIN_NSPEC_INNER_CORE
#define COLORING_MIN_NSPEC_INNER_CORE 1000
#endif
#ifndef COLORING_MIN_NSPEC_OUTER_CORE
#define COLORING_MIN_NSPEC_OUTER_CORE 1000
#endif
#ifndef BLOCKSIZE_TRANSFER
#define BLOCKSIZE_TRANSFER 256
#endif
__global__ void compute_coupling_fluid_CMB_kernel(const float * displ_crust_mantle, float * accel_outer_core, const int * ibool_crust_mantle, const int * ibelm_bottom_crust_mantle, const float * normal_top_outer_core, const float * jacobian2D_top_outer_core, const float * wgllwgll_xy, const int * ibool_outer_core, const int * ibelm_top_outer_core, const int NSPEC2D_TOP_OC){
  int i;
  int j;
  int k;
  int iface;
  int k_corresp;
  float displ_n;
  int iglob_cm;
  int iglob_oc;
  int ispec;
  int ispec_selected;
  float displ_x;
  float displ_y;
  float displ_z;
  float nx;
  float ny;
  float nz;
  float weight;
  i = threadIdx.x;
  j = threadIdx.y;
  iface = blockIdx.x + (gridDim.x) * (blockIdx.y);
  if(iface < NSPEC2D_TOP_OC){
    ispec = ibelm_top_outer_core[iface - 0] - (1);
    ispec_selected = ibelm_bottom_crust_mantle[iface - 0] - (1);
    k = NGLLX - (1);
    k_corresp = 0;
    iglob_cm = ibool_crust_mantle[INDEX4(NGLLX, NGLLX, NGLLX, i, j, k_corresp, ispec_selected) - 0] - (1);
    displ_x = displ_crust_mantle[(iglob_cm) * (3) + 0 - 0];
    displ_y = displ_crust_mantle[(iglob_cm) * (3) + 1 - 0];
    displ_z = displ_crust_mantle[(iglob_cm) * (3) + 2 - 0];
    nx = normal_top_outer_core[INDEX4(NDIM, NGLLX, NGLLX, 0, i, j, iface) - 0];
    ny = normal_top_outer_core[INDEX4(NDIM, NGLLX, NGLLX, 1, i, j, iface) - 0];
    nz = normal_top_outer_core[INDEX4(NDIM, NGLLX, NGLLX, 2, i, j, iface) - 0];
    displ_n = (displ_x) * (nx) + (displ_y) * (ny) + (displ_z) * (nz);
    weight = (jacobian2D_top_outer_core[INDEX3(NGLLX, NGLLX, i, j, iface) - 0]) * (wgllwgll_xy[INDEX2(NGLLX, i, j) - 0]);
    iglob_oc = ibool_outer_core[INDEX4(NGLLX, NGLLX, NGLLX, i, j, k, ispec) - 0] - (1);
    atomicAdd(accel_outer_core + iglob_oc, (weight) * (displ_n));
  }
}