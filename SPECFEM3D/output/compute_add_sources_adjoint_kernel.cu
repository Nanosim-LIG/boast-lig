
#include <hip/hip_runtime.h>
#ifndef INDEX2
#define INDEX2(isize,i,j) i + isize*j
#endif
#ifndef INDEX3
#define INDEX3(isize,jsize,i,j,k) i + isize*(j + jsize*k)
#endif
#ifndef INDEX4
#define INDEX4(isize,jsize,ksize,i,j,k,x) i + isize*(j + jsize*(k + ksize*x))
#endif
#ifndef INDEX5
#define INDEX5(isize,jsize,ksize,xsize,i,j,k,x,y) i + isize*(j + jsize*(k + ksize*(x + xsize*y)))
#endif
#ifndef NDIM
#define NDIM 3
#endif
#ifndef NGLLX
#define NGLLX 5
#endif
#ifndef NGLL2
#define NGLL2 25
#endif
#ifndef NGLL3
#define NGLL3 125
#endif
#ifndef NGLL3_PADDED
#define NGLL3_PADDED 128
#endif
#ifndef N_SLS
#define N_SLS 3
#endif
#ifndef IREGION_CRUST_MANTLE
#define IREGION_CRUST_MANTLE 1
#endif
#ifndef IREGION_INNER_CORE
#define IREGION_INNER_CORE 3
#endif
#ifndef IFLAG_IN_FICTITIOUS_CUBE
#define IFLAG_IN_FICTITIOUS_CUBE 11
#endif
#ifndef R_EARTH_KM
#define R_EARTH_KM 6371.0f
#endif
#ifndef COLORING_MIN_NSPEC_INNER_CORE
#define COLORING_MIN_NSPEC_INNER_CORE 1000
#endif
#ifndef COLORING_MIN_NSPEC_OUTER_CORE
#define COLORING_MIN_NSPEC_OUTER_CORE 1000
#endif
#ifndef BLOCKSIZE_TRANSFER
#define BLOCKSIZE_TRANSFER 256
#endif
__global__ void compute_add_sources_adjoint_kernel(float * accel, const int nrec, const float * adj_sourcearrays, const int * ibool, const int * ispec_selected_rec, const int * pre_computed_irec, const int nadj_rec_local){
  int ispec;
  int iglob;
  int irec_local;
  int irec;
  int i;
  int j;
  int k;
  irec_local = blockIdx.x + (gridDim.x) * (blockIdx.y);
  if (irec_local < nadj_rec_local) {
    irec = pre_computed_irec[irec_local];
    ispec = ispec_selected_rec[irec] - (1);
    i = threadIdx.x;
    j = threadIdx.y;
    k = threadIdx.z;
    iglob = ibool[INDEX4(NGLLX, NGLLX, NGLLX, i, j, k, ispec)] - (1);
    atomicAdd(accel + (iglob) * (3) + 0, adj_sourcearrays[INDEX5(NDIM, NGLLX, NGLLX, NGLLX, 0, i, j, k, irec_local)]);
    atomicAdd(accel + (iglob) * (3) + 1, adj_sourcearrays[INDEX5(NDIM, NGLLX, NGLLX, NGLLX, 1, i, j, k, irec_local)]);
    atomicAdd(accel + (iglob) * (3) + 2, adj_sourcearrays[INDEX5(NDIM, NGLLX, NGLLX, NGLLX, 2, i, j, k, irec_local)]);
  }
}
