
#include <hip/hip_runtime.h>
#ifndef INDEX2
#define INDEX2(isize,i,j) i + isize*j
#endif
#ifndef INDEX3
#define INDEX3(isize,jsize,i,j,k) i + isize*(j + jsize*k)
#endif
#ifndef INDEX4
#define INDEX4(isize,jsize,ksize,i,j,k,x) i + isize*(j + jsize*(k + ksize*x))
#endif
#ifndef INDEX5
#define INDEX5(isize,jsize,ksize,xsize,i,j,k,x,y) i + isize*(j + jsize*(k + ksize*(x + xsize*y)))
#endif
#ifndef NDIM
#define NDIM 3
#endif
#ifndef NGLLX
#define NGLLX 5
#endif
#ifndef NGLL2
#define NGLL2 25
#endif
#ifndef NGLL3
#define NGLL3 125
#endif
#ifndef NGLL3_PADDED
#define NGLL3_PADDED 128
#endif
#ifndef N_SLS
#define N_SLS 3
#endif
#ifndef IREGION_CRUST_MANTLE
#define IREGION_CRUST_MANTLE 1
#endif
#ifndef IREGION_INNER_CORE
#define IREGION_INNER_CORE 3
#endif
#ifndef IFLAG_IN_FICTITIOUS_CUBE
#define IFLAG_IN_FICTITIOUS_CUBE 11
#endif
#ifndef R_EARTH_KM
#define R_EARTH_KM 6371.0f
#endif
#ifndef COLORING_MIN_NSPEC_INNER_CORE
#define COLORING_MIN_NSPEC_INNER_CORE 1000
#endif
#ifndef COLORING_MIN_NSPEC_OUTER_CORE
#define COLORING_MIN_NSPEC_OUTER_CORE 1000
#endif
#ifndef BLOCKSIZE_TRANSFER
#define BLOCKSIZE_TRANSFER 256
#endif
__global__ void write_seismograms_transfer_strain_from_device_kernel(const int * number_receiver_global, const int * ispec_selected_rec, const int * ibool, float * station_strain_field, const float * d_field, const int nrec_local){
  int tx;
  int irec;
  int ispec;
  int iglob;
  int blockID;
  blockID = blockIdx.x + (blockIdx.y) * (gridDim.x);
  tx = threadIdx.x;
  if (blockID < nrec_local) {
    irec = number_receiver_global[blockID] - (1);
    ispec = ispec_selected_rec[irec] - (1);
    iglob = ibool[tx + (NGLL3) * (ispec)] - (1);
    station_strain_field[(NGLL3) * (blockID) + tx] = d_field[iglob];
  }
}
