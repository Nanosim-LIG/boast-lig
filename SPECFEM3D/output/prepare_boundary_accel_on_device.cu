
#include <hip/hip_runtime.h>
#ifndef INDEX2
#define INDEX2(xsize,x,y) x + (y)*xsize
#endif
#ifndef INDEX3
#define INDEX3(xsize,ysize,x,y,z) x + xsize*(y + ysize*z)
#endif
#ifndef INDEX4
#define INDEX4(xsize,ysize,zsize,x,y,z,i) x + xsize*(y + ysize*(z + zsize*i))
#endif
#ifndef INDEX5
#define INDEX5(xsize,ysize,zsize,isize,x,y,z,i,j) x + xsize*(y + ysize*(z + zsize*(i + isize*(j))))
#endif
#ifndef NDIM
#define NDIM 3
#endif
#ifndef NGLLX
#define NGLLX 5
#endif
#ifndef NGLL2
#define NGLL2 25
#endif
#ifndef NGLL3
#define NGLL3 125
#endif
#ifndef NGLL3_PADDED
#define NGLL3_PADDED 128
#endif
#ifndef N_SLS
#define N_SLS 3
#endif
#ifndef IREGION_CRUST_MANTLE
#define IREGION_CRUST_MANTLE 1
#endif
#ifndef IREGION_INNER_CORE
#define IREGION_INNER_CORE 3
#endif
#ifndef IFLAG_IN_FICTITIOUS_CUBE
#define IFLAG_IN_FICTITIOUS_CUBE 11
#endif
#ifndef R_EARTH_KM
#define R_EARTH_KM 6371.0f
#endif
#ifndef COLORING_MIN_NSPEC_INNER_CORE
#define COLORING_MIN_NSPEC_INNER_CORE 1000
#endif
#ifndef COLORING_MIN_NSPEC_OUTER_CORE
#define COLORING_MIN_NSPEC_OUTER_CORE 1000
#endif
#ifndef BLOCKSIZE_TRANSFER
#define BLOCKSIZE_TRANSFER 256
#endif
__global__ void prepare_boundary_accel_on_device(const float * d_accel, float * d_send_accel_buffer, const int num_interfaces, const int max_nibool_interfaces, const int * d_nibool_interfaces, const int * d_ibool_interfaces){
  int id;
  int iglob;
  int iloc;
  int iinterface;
  id = threadIdx.x + (blockIdx.x) * (blockDim.x) + ((gridDim.x) * (blockDim.x)) * (threadIdx.y + (blockIdx.y) * (blockDim.y));
  for(iinterface=0; iinterface<=num_interfaces - (1); iinterface+=1){
    if(id < d_nibool_interfaces[iinterface - 0]){
      iloc = id + (max_nibool_interfaces) * (iinterface);
      iglob = d_ibool_interfaces[iloc - 0] - (1);
      d_send_accel_buffer[(iloc) * (3) + 0 - 0] = d_accel[(iglob) * (3) + 0 - 0];
      d_send_accel_buffer[(iloc) * (3) + 1 - 0] = d_accel[(iglob) * (3) + 1 - 0];
      d_send_accel_buffer[(iloc) * (3) + 2 - 0] = d_accel[(iglob) * (3) + 2 - 0];
    }
  }
}