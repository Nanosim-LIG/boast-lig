
#include <hip/hip_runtime.h>
#ifndef INDEX2
#define INDEX2(isize,i,j) i + isize*j
#endif
#ifndef INDEX3
#define INDEX3(isize,jsize,i,j,k) i + isize*(j + jsize*k)
#endif
#ifndef INDEX4
#define INDEX4(isize,jsize,ksize,i,j,k,x) i + isize*(j + jsize*(k + ksize*x))
#endif
#ifndef INDEX5
#define INDEX5(isize,jsize,ksize,xsize,i,j,k,x,y) i + isize*(j + jsize*(k + ksize*(x + xsize*y)))
#endif
#ifndef NDIM
#define NDIM 3
#endif
#ifndef NGLLX
#define NGLLX 5
#endif
#ifndef NGLL2
#define NGLL2 25
#endif
#ifndef NGLL3
#define NGLL3 125
#endif
#ifndef NGLL3_PADDED
#define NGLL3_PADDED 128
#endif
#ifndef N_SLS
#define N_SLS 3
#endif
#ifndef IREGION_CRUST_MANTLE
#define IREGION_CRUST_MANTLE 1
#endif
#ifndef IREGION_INNER_CORE
#define IREGION_INNER_CORE 3
#endif
#ifndef IFLAG_IN_FICTITIOUS_CUBE
#define IFLAG_IN_FICTITIOUS_CUBE 11
#endif
#ifndef R_EARTH_KM
#define R_EARTH_KM 6371.0f
#endif
#ifndef COLORING_MIN_NSPEC_INNER_CORE
#define COLORING_MIN_NSPEC_INNER_CORE 1000
#endif
#ifndef COLORING_MIN_NSPEC_OUTER_CORE
#define COLORING_MIN_NSPEC_OUTER_CORE 1000
#endif
#ifndef BLOCKSIZE_TRANSFER
#define BLOCKSIZE_TRANSFER 256
#endif
__global__ void noise_add_source_master_rec_kernel(const int * ibool, const int * ispec_selected_rec, const int irec_master_noise, float * accel, const float * noise_sourcearray, const int it){
  int tx;
  int ispec;
  int iglob;
  tx = threadIdx.x;
  ispec = ispec_selected_rec[irec_master_noise] - (1);
  iglob = ibool[tx + (NGLL3) * (ispec)] - (1);
  atomicAdd(accel + (iglob) * (3) + 0, noise_sourcearray[(tx) * (3) + ((NGLL3) * (3)) * (it) + 0]);
  atomicAdd(accel + (iglob) * (3) + 1, noise_sourcearray[(tx) * (3) + ((NGLL3) * (3)) * (it) + 1]);
  atomicAdd(accel + (iglob) * (3) + 2, noise_sourcearray[(tx) * (3) + ((NGLL3) * (3)) * (it) + 2]);
}
