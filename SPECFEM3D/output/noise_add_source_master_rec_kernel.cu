
#include <hip/hip_runtime.h>
#ifndef INDEX2
#define INDEX2(xsize,x,y) x + (y)*xsize
#endif
#ifndef INDEX3
#define INDEX3(xsize,ysize,x,y,z) x + xsize*(y + ysize*z)
#endif
#ifndef INDEX4
#define INDEX4(xsize,ysize,zsize,x,y,z,i) x + xsize*(y + ysize*(z + zsize*i))
#endif
#ifndef INDEX5
#define INDEX5(xsize,ysize,zsize,isize,x,y,z,i,j) x + xsize*(y + ysize*(z + zsize*(i + isize*(j))))
#endif
#ifndef NDIM
#define NDIM 3
#endif
#ifndef NGLLX
#define NGLLX 5
#endif
#ifndef NGLL2
#define NGLL2 25
#endif
#ifndef NGLL3
#define NGLL3 125
#endif
#ifndef NGLL3_PADDED
#define NGLL3_PADDED 128
#endif
#ifndef N_SLS
#define N_SLS 3
#endif
#ifndef IREGION_CRUST_MANTLE
#define IREGION_CRUST_MANTLE 1
#endif
#ifndef IREGION_INNER_CORE
#define IREGION_INNER_CORE 3
#endif
#ifndef IFLAG_IN_FICTITIOUS_CUBE
#define IFLAG_IN_FICTITIOUS_CUBE 11
#endif
#ifndef R_EARTH_KM
#define R_EARTH_KM 6371.0f
#endif
#ifndef COLORING_MIN_NSPEC_INNER_CORE
#define COLORING_MIN_NSPEC_INNER_CORE 1000
#endif
#ifndef COLORING_MIN_NSPEC_OUTER_CORE
#define COLORING_MIN_NSPEC_OUTER_CORE 1000
#endif
#ifndef BLOCKSIZE_TRANSFER
#define BLOCKSIZE_TRANSFER 256
#endif
__global__ void noise_add_source_master_rec_kernel(const int * ibool, const int * ispec_selected_rec, const int irec_master_noise, float * accel, const float * noise_sourcearray, const int it){
  int tx;
  int ispec;
  int iglob;
  tx = threadIdx.x;
  ispec = ispec_selected_rec[irec_master_noise - 0] - (1);
  iglob = ibool[tx + (NGLL3) * (ispec) - 0] - (1);
  atomicAdd(accel + (iglob) * (3) + 0, noise_sourcearray[(tx) * (3) + ((NGLL3) * (3)) * (it) + 0 - 0]);
  atomicAdd(accel + (iglob) * (3) + 1, noise_sourcearray[(tx) * (3) + ((NGLL3) * (3)) * (it) + 1 - 0]);
  atomicAdd(accel + (iglob) * (3) + 2, noise_sourcearray[(tx) * (3) + ((NGLL3) * (3)) * (it) + 2 - 0]);
}