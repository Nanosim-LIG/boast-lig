
#include <hip/hip_runtime.h>
#ifndef INDEX2
#define INDEX2(isize,i,j) i + isize*j
#endif
#ifndef INDEX3
#define INDEX3(isize,jsize,i,j,k) i + isize*(j + jsize*k)
#endif
#ifndef INDEX4
#define INDEX4(isize,jsize,ksize,i,j,k,x) i + isize*(j + jsize*(k + ksize*x))
#endif
#ifndef INDEX5
#define INDEX5(isize,jsize,ksize,xsize,i,j,k,x,y) i + isize*(j + jsize*(k + ksize*(x + xsize*y)))
#endif
#ifndef NDIM
#define NDIM 3
#endif
#ifndef NGLLX
#define NGLLX 5
#endif
#ifndef NGLL2
#define NGLL2 25
#endif
#ifndef NGLL3
#define NGLL3 125
#endif
#ifndef NGLL3_PADDED
#define NGLL3_PADDED 128
#endif
#ifndef N_SLS
#define N_SLS 3
#endif
#ifndef IREGION_CRUST_MANTLE
#define IREGION_CRUST_MANTLE 1
#endif
#ifndef IREGION_INNER_CORE
#define IREGION_INNER_CORE 3
#endif
#ifndef IFLAG_IN_FICTITIOUS_CUBE
#define IFLAG_IN_FICTITIOUS_CUBE 11
#endif
#ifndef R_EARTH_KM
#define R_EARTH_KM 6371.0f
#endif
#ifndef COLORING_MIN_NSPEC_INNER_CORE
#define COLORING_MIN_NSPEC_INNER_CORE 1000
#endif
#ifndef COLORING_MIN_NSPEC_OUTER_CORE
#define COLORING_MIN_NSPEC_OUTER_CORE 1000
#endif
#ifndef BLOCKSIZE_TRANSFER
#define BLOCKSIZE_TRANSFER 256
#endif
__global__ void update_accel_elastic_kernel(float * accel, const float * veloc, const int size, const float two_omega_earth, const float * rmassx, const float * rmassy, const float * rmassz){
  int id;
  id = threadIdx.x + (blockIdx.x) * (blockDim.x) + (blockIdx.y) * ((gridDim.x) * (blockDim.x));
  if (id < size) {
    accel[(id) * (3)] = (accel[(id) * (3)]) * (rmassx[id]) + (two_omega_earth) * (veloc[(id) * (3) + 1]);
    accel[(id) * (3) + 1] = (accel[(id) * (3) + 1]) * (rmassy[id]) - ((two_omega_earth) * (veloc[(id) * (3)]));
    accel[(id) * (3) + 2] = (accel[(id) * (3) + 2]) * (rmassz[id]);
  }
}
